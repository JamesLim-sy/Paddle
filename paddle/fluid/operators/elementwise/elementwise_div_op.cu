#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/elementwise/elementwise_div_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/platform/complex128.h"
#include "paddle/fluid/platform/complex64.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename T, typename Enable = void>
struct CudaDivFunctor {
  inline HOSTDEVICE T operator()(const T* args) const {
    return args[0] / args[1];
  }
};

template <typename T>
struct CudaDivFunctor<T,
                      typename std::enable_if_t<std::is_integral<T>::value>> {
  inline HOSTDEVICE T operator()(const T* args) const {
    PADDLE_ENFORCE(args[1] != 0,
                   "InvalidArgumentError: Integer division "
                   "by zero encountered in divide. Please check.\n");
    return args[0] / args[1];
  }
};

template <typename T>
class ElementwiseDivKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    std::vector<const framework::Tensor*> ins;
    std::vector<framework::Tensor*> outs;
    PackTensorsIntoVector<T>(ctx, &ins, &outs);
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        ctx, ins, &outs, CudaDivFunctor<T>());
  }
};

template <typename T>
static __global__ void SimpleElemwiseDivGradCUDAKernel(const T* x, const T* y,
                                                       const T* out,
                                                       const T* dout,
                                                       int64_t size, T* dx,
                                                       T* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    T o = dout[col];
    dx[col] = o / y[col];
    dy[col] = -o * out[col] / y[col];
    col += blockDim.x * gridDim.x;
  }
}

template <>
__global__ void SimpleElemwiseDivGradCUDAKernel<paddle::platform::complex64>(
    const paddle::platform::complex64* x, const paddle::platform::complex64* y,
    const paddle::platform::complex64* out,
    const paddle::platform::complex64* dout, int64_t size,
    paddle::platform::complex64* dx, paddle::platform::complex64* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    paddle::platform::complex64 o = dout[col];
    paddle::platform::complex64 y_conj(y[col].real, -y[col].imag);
    paddle::platform::complex64 out_div_y_conj((out[col] / y[col]).real,
                                               -(out[col] / y[col]).imag);
    dx[col] = o / y_conj;
    dy[col] = -o * out_div_y_conj;
    col += blockDim.x * gridDim.x;
  }
}

template <>
__global__ void SimpleElemwiseDivGradCUDAKernel<paddle::platform::complex128>(
    const paddle::platform::complex128* x,
    const paddle::platform::complex128* y,
    const paddle::platform::complex128* out,
    const paddle::platform::complex128* dout, int64_t size,
    paddle::platform::complex128* dx, paddle::platform::complex128* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    paddle::platform::complex128 o = dout[col];
    paddle::platform::complex128 y_conj(y[col].real, -y[col].imag);
    paddle::platform::complex128 out_div_y_conj((out[col] / y[col]).real,
                                                -(out[col] / y[col]).imag);
    dx[col] = o / y_conj;
    dy[col] = -o * out_div_y_conj;
    col += blockDim.x * gridDim.x;
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, plat::CUDADeviceContext>::value>::type
elementwise_div_grad(const framework::ExecutionContext& ctx,
                     const framework::Tensor* x, const framework::Tensor* y,
                     const framework::Tensor* out,
                     const framework::Tensor* dout, framework::Tensor* dx,
                     framework::Tensor* dy) {
  dim3 block_size = dim3(PADDLE_CUDA_THREAD_SIZE, 1);
  auto size = x->numel();
  dim3 grid_size =
      dim3((size + PADDLE_CUDA_THREAD_SIZE - 1) / PADDLE_CUDA_THREAD_SIZE, 1);
  SimpleElemwiseDivGradCUDAKernel<
      T><<<grid_size, block_size, 0,
           ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
      x->data<T>(), y->data<T>(), out->data<T>(), dout->data<T>(), size,
      dx->mutable_data<T>(ctx.GetPlace()), dy->mutable_data<T>(ctx.GetPlace()));
}

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_div,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::float16>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex64>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex128>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_div_grad,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex64>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex128>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_div_grad_grad,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::float16>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        double>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int64_t>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex64>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex128>);
