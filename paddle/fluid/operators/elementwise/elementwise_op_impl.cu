#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.1 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.1
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#pragma once

#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/device_context.h"
#include "paddle/fluid/platform/fast_divmod.h"
#include "paddle/fluid/platform/bfloat16.h"

namespace plat = paddle::platform;

#ifdef __HIPCC__
#define ELEMENTWISE_BLOCK_SIZE 256
#else
#define ELEMENTWISE_BLOCK_SIZE 512
#endif

namespace paddle {
namespace operators {

/*
* According to NVIDIA, if number of threads per block is 64/128/256/512,
* cuda performs better. And number of blocks should be greater (at least
* 2x~4x) than number of SMs. Hence, SM count is took into account within
* this function to determine the right number of threads per block.
*/
int GetThreadsConfig(const platform::CUDADeviceContext &ctx,
                            int64_t numel, int vec_size) {
  int threads = ELEMENTWISE_BLOCK_SIZE;
  int sm_count = ctx.GetSMCount();
  int active_threads_num = numel / vec_size;
  if (active_threads_num / (sm_count << 1) < ELEMENTWISE_BLOCK_SIZE) {
    // Round up threads number into an exponential multiple of 2, while number
    // of acitve blocks is about twice of SM, to acquire better performance.
    threads = platform::RoundToPowerOfTwo(active_threads_num / (sm_count << 1));
  } else if (active_threads_num / (sm_count << 2) < ELEMENTWISE_BLOCK_SIZE) {
    // Round up threads number into an exponential multiple of 2, while number
    // of acitve blocks is about 4 times of SM, to acquire better performance.
    threads = platform::RoundToPowerOfTwo(active_threads_num / (sm_count << 2));
  }
  // Number of threads per block shall be larger than 64.
  return std::max(64, threads);
}

/*
* Only the address of input data is the multiplier of 1,2,4, vectorized load
* with corresponding multiplier-value is possible. Moreover, the maximum length
* of vectorized load is 128 bits once. Hence, valid length of vectorized load
* shall be determined under both former constraints.
*/
template <typename T>
int GetVectorizedSizeImpl(const T *pointer) {
  constexpr int max_load_bits = 128;
  int valid_vec_size = max_load_bits / CHAR_BIT / sizeof(T);
  uint64_t address = reinterpret_cast<uint64_t>(pointer);
  constexpr int vec8 =
      std::alignment_of<CudaAlignedVector<T, 8>>::value;  // NOLINT
  constexpr int vec4 =
      std::alignment_of<CudaAlignedVector<T, 4>>::value;  // NOLINT
  constexpr int vec2 =
      std::alignment_of<CudaAlignedVector<T, 2>>::value;  // NOLINT
  if (address % vec8 == 0) {
    /*
    * Currently, decide to deal with no more than 4 data once while adopting
    * vectorization load/store, if performance test shows that dealing with
    * 8 data once in vectorization load/store does get optimized, return code
    * below can be changed into " return std::min(8, valid_vec_size); " .
    */
    return std::min(8, valid_vec_size);
  } else if (address % vec4 == 0) {
    return std::min(4, valid_vec_size);
  } else if (address % vec2 == 0) {
    return std::min(2, valid_vec_size);
  } else {
    return 1;
  }
}

template int GetVectorizedSizeImpl(const bool *pointer);
template int GetVectorizedSizeImpl(const signed char *pointer);
template int GetVectorizedSizeImpl(const unsigned char *pointer);
template int GetVectorizedSizeImpl(const short *pointer);
template int GetVectorizedSizeImpl(const int *pointer);
template int GetVectorizedSizeImpl(const int64_t *pointer);
template int GetVectorizedSizeImpl(const float *pointer);
template int GetVectorizedSizeImpl(const double *pointer);
template int GetVectorizedSizeImpl(const plat::float16 *pointer);
template int GetVectorizedSizeImpl(const plat::bfloat16 *pointer);
template int GetVectorizedSizeImpl(const plat::complex<float> *pointer);
template int GetVectorizedSizeImpl(const plat::complex<double> *pointer);

}  // namespace operators
}  // namespace paddle
